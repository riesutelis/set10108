#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <vector>

using namespace std;

__global__ void simple_multiply(float *output_C, unsigned int width_A, unsigned int height_A, unsigned int width_B, unsigned int height_B, const float *input_A, const float *input_B)
{
	// Get global position in Y direction
	//unsigned int row = (blockIdx.y * 16) + threadIdx.y;
	// Get global position in X direction
	//unsigned int col = (blockIdx.x * 16) + threadIdx.x;
	unsigned int row = threadIdx.x;
	// Get global position in X direction
	unsigned int col = blockIdx.x;

	float sum = 0.0f;

	// Calculate result of one element of matrix C
	for (unsigned int i = 0; i < width_A; ++i)
		sum += input_A[row * width_A + i] * input_B[i * width_B + col];

	// Store result in matrix C
	output_C[row * width_B + col] = sum;
}

int main(int argc, char **argv)
{
	unsigned int A_Width = 16, B_Width = 16, A_Height = 16, B_Height = 16;
	auto size_A = sizeof(float) * A_Width * A_Height;
	auto size_B = sizeof(float) * B_Width * B_Height;
	auto size_C = sizeof(float) * B_Width * A_Height;
	vector<float> A(A_Width * A_Height);
	vector<float> B(B_Width * B_Height);
	vector<float> C(A_Height * B_Width);

	for (int i = 0; i < A_Width; i++)
		for (int j = 0; j < A_Height; j++)
			A[i * A_Width + j] = 1.0f;
	for (int i = 0; i < B_Width; i++)
		for (int j = 0; j < B_Height; j++)
			B[i * B_Width + j] = 1.0f;

	// Device memory
	float *d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A, size_A);
	hipMalloc((void**)&d_B, size_B);
	hipMalloc((void**)&d_C, size_C);

	hipMemcpy(d_A, &A[0], size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, &B[0], size_B, hipMemcpyHostToDevice);

	simple_multiply <<<A_Height, B_Width >> > (d_C, A_Width, A_Height, B_Width, B_Height, d_A, d_B);
	hipDeviceSynchronize();

	hipMemcpy(&C[0], d_C, size_C, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	for (int i = 0; i < A_Height; i++)
	{
		for (int j = 0; j < B_Width; j++)
			cout << C[A_Height * i + j] << " ";
		cout << endl;
	}

	return 0;
}